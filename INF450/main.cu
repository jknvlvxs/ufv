// #@title Código da Main (parcialmente Genérico, 64 blocos...)
// %%writefile main.cu

int main(int argc, char **argv) {
    printf("%s Starting...\n", argv[0]);
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int Dim = 3;
    int k = 15;
    int nElem = (1 << POWER) * Dim;
    printf("Vector Size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);
    size_t cBytes = Dim * k * sizeof(float);
    size_t tBytes = k * sizeof(int);
    size_t vBytes = GRID * (k * Dim + k) * sizeof(float); // 64 blks, 60 variables k=15, d=3

    float *h_data, *h_centroids;
    float *h_v_r;
    h_data = (float *)malloc(nBytes);
    h_v_r = (float *)malloc(vBytes);
    initialData(h_data, nElem);
    h_centroids = (float *)malloc(cBytes);
    h_centroids[0] = 100;
    h_centroids[1] = 100;
    h_centroids[2] = 100;
    h_centroids[3] = 200;
    h_centroids[4] = 200;
    h_centroids[5] = 200;
    h_centroids[6] = 300;
    h_centroids[7] = 300;
    h_centroids[8] = 300;
    h_centroids[9] = 400;
    h_centroids[10] = 400;
    h_centroids[11] = 400;
    h_centroids[12] = 100;
    h_centroids[13] = 100;
    h_centroids[14] = 100;
    h_centroids[15] = 200;
    h_centroids[16] = 200;
    h_centroids[17] = 200;
    h_centroids[18] = 300;
    h_centroids[19] = 300;
    h_centroids[20] = 300;
    h_centroids[21] = 400;
    h_centroids[22] = 400;
    h_centroids[23] = 400;
    h_centroids[24] = 500;
    h_centroids[25] = 500;
    h_centroids[26] = 500;
    h_centroids[27] = 100;
    h_centroids[28] = 100;
    h_centroids[29] = 100;
    h_centroids[30] = 200;
    h_centroids[31] = 200;
    h_centroids[32] = 200;
    h_centroids[33] = 300;
    h_centroids[34] = 300;
    h_centroids[35] = 300;
    h_centroids[36] = 400;
    h_centroids[37] = 400;
    h_centroids[38] = 400;
    h_centroids[39] = 500;
    h_centroids[40] = 500;
    h_centroids[41] = 500;
    h_centroids[42] = 600;
    h_centroids[43] = 600;
    h_centroids[44] = 600;

    float *h_counter;
    h_counter = (float *)malloc(tBytes);

    h_counter[0] = 0;
    h_counter[1] = 0;
    h_counter[2] = 0;
    h_counter[3] = 0;
    h_counter[4] = 0;
    h_counter[5] = 0;
    h_counter[6] = 0;
    h_counter[7] = 0;
    h_counter[8] = 0;
    h_counter[9] = 0;
    h_counter[10] = 0;
    h_counter[11] = 0;
    h_counter[12] = 0;
    h_counter[13] = 0;
    h_counter[14] = 0;

    // malloc device global memory
    float *d_data, *d_centroids;
    float *v_r;
    CHECK(hipMalloc((float **)&v_r, vBytes));
    CHECK(hipMalloc((float **)&d_data, nBytes));
    CHECK(hipMalloc((float **)&d_centroids, cBytes));
    int *d_counter;
    CHECK(hipMalloc((int **)&d_counter, tBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_data, h_data, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_centroids, h_centroids, cBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_counter, h_counter, tBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int iLen = BLOCO;
    dim3 block(iLen);
    // dim3 grid  (((nElem/Dim + block.x - 1) / block.x));
    dim3 grid2(GRID);

    for (int i = 0; i < 8; i++) {
        kmeansReduce<<<grid2, block>>>(d_data, d_centroids, d_counter, nElem, v_r);
        kernel<<<1, GRID / 2>>>(v_r); // So foi testado com grid=64, 32 threads
        hipDeviceSynchronize();
        CHECK(hipMemcpy(h_v_r, v_r, vBytes, hipMemcpyDeviceToHost));
        h_centroids[0] = h_v_r[0] / h_v_r[4 * GRID];
        h_centroids[1] = h_v_r[GRID] / h_v_r[4 * GRID];
        h_centroids[2] = h_v_r[GRID * 2] / h_v_r[4 * GRID];
        h_centroids[3] = h_v_r[GRID * 3] / h_v_r[5 * GRID];
        h_centroids[4] = h_v_r[GRID * 4] / h_v_r[5 * GRID];
        h_centroids[5] = h_v_r[GRID * 5] / h_v_r[5 * GRID];
        h_centroids[6] = h_v_r[GRID * 6] / h_v_r[6 * GRID];
        h_centroids[7] = h_v_r[GRID * 7] / h_v_r[6 * GRID];
        h_centroids[8] = h_v_r[GRID * 8] / h_v_r[6 * GRID];
        h_centroids[9] = h_v_r[GRID * 9] / h_v_r[7 * GRID];
        h_centroids[10] = h_v_r[GRID * 10] / h_v_r[7 * GRID];
        h_centroids[11] = h_v_r[GRID * 11] / h_v_r[7 * GRID];
        h_centroids[12] = h_v_r[GRID * 12] / h_v_r[8 * GRID];
        h_centroids[13] = h_v_r[GRID * 13] / h_v_r[8 * GRID];
        h_centroids[14] = h_v_r[GRID * 14] / h_v_r[8 * GRID];
        h_centroids[15] = h_v_r[GRID * 15] / h_v_r[9 * GRID];
        h_centroids[16] = h_v_r[GRID * 16] / h_v_r[9 * GRID];
        h_centroids[17] = h_v_r[GRID * 17] / h_v_r[9 * GRID];
        h_centroids[18] = h_v_r[GRID * 18] / h_v_r[10 * GRID];
        h_centroids[19] = h_v_r[GRID * 19] / h_v_r[10 * GRID];
        h_centroids[20] = h_v_r[GRID * 20] / h_v_r[10 * GRID];
        h_centroids[21] = h_v_r[GRID * 21] / h_v_r[11 * GRID];
        h_centroids[22] = h_v_r[GRID * 22] / h_v_r[11 * GRID];
        h_centroids[23] = h_v_r[GRID * 23] / h_v_r[11 * GRID];
        h_centroids[24] = h_v_r[GRID * 24] / h_v_r[12 * GRID];
        h_centroids[25] = h_v_r[GRID * 25] / h_v_r[12 * GRID];
        h_centroids[26] = h_v_r[GRID * 26] / h_v_r[12 * GRID];
        h_centroids[27] = h_v_r[GRID * 27] / h_v_r[13 * GRID];
        h_centroids[28] = h_v_r[GRID * 28] / h_v_r[13 * GRID];
        h_centroids[29] = h_v_r[GRID * 29] / h_v_r[13 * GRID];
        h_centroids[30] = h_v_r[GRID * 30] / h_v_r[14 * GRID];
        h_centroids[31] = h_v_r[GRID * 31] / h_v_r[14 * GRID];
        h_centroids[32] = h_v_r[GRID * 32] / h_v_r[14 * GRID];
        h_centroids[33] = h_v_r[GRID * 33] / h_v_r[15 * GRID];
        h_centroids[34] = h_v_r[GRID * 34] / h_v_r[15 * GRID];
        h_centroids[35] = h_v_r[GRID * 35] / h_v_r[15 * GRID];
        h_centroids[36] = h_v_r[GRID * 36] / h_v_r[16 * GRID];
        h_centroids[37] = h_v_r[GRID * 37] / h_v_r[16 * GRID];
        h_centroids[38] = h_v_r[GRID * 38] / h_v_r[16 * GRID];
        h_centroids[39] = h_v_r[GRID * 39] / h_v_r[17 * GRID];
        h_centroids[40] = h_v_r[GRID * 40] / h_v_r[17 * GRID];
        h_centroids[41] = h_v_r[GRID * 41] / h_v_r[17 * GRID];
        h_centroids[42] = h_v_r[GRID * 42] / h_v_r[18 * GRID];
        h_centroids[43] = h_v_r[GRID * 43] / h_v_r[18 * GRID];
        h_centroids[44] = h_v_r[GRID * 44] / h_v_r[18 * GRID];

        for (int j = 0; j < k * Dim; j += Dim) {
            printf(" centroids dim %d value %f  \n", j, h_centroids[j]);
        }
        CHECK(hipDeviceSynchronize());
        // check kernel error
        CHECK(hipGetLastError());
        memset(h_counter, 0, tBytes);
        CHECK(hipMemcpy(d_centroids, h_centroids, cBytes, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_counter, h_counter, tBytes, hipMemcpyHostToDevice));
    }
    // check device results

    // free device global memory
    CHECK(hipFree(d_data));
    CHECK(hipFree(d_centroids));
    CHECK(hipFree(v_r));
    CHECK(hipFree(d_counter));

    // free host memory
    free(h_data);
    free(h_centroids);
    free(h_v_r);
    free(h_counter);

    return (0);
}