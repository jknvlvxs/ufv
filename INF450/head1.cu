#include <sys/time.h>
#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess) {                                \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
        }                                                          \
    }

#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * kmeans example 3D
 */

void initialData(float *ip, int size) {
    int i;
    for (i = 0; i < size; i += 3) {
        ip[i] = i;
        ip[i + 1] = i;
        ip[i + 2] = i;
    }
    printf("maior valor %d/n", i);
    return;
}
