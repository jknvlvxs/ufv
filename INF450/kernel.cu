#include "hip/hip_runtime.h"
// #@title Kernel principal e Kernel de Redução - Fixo em 64 numero de blocos
// %%writefile kernel.cu

__global__ void kernel(float *v_r) {

    int stride = gridDim.x * 2;
    for (int s = stride / 2; s > 0; s >>= 1) {

        if (threadIdx.x < s) {
            v_r[threadIdx.x] += v_r[threadIdx.x + s];
            v_r[threadIdx.x + stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 2 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 3 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 4 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 5 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 6 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 7 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 8 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 9 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 10 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 11 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 12 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 13 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 14 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 15 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 16 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 17 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 18 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 19 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 20 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 21 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 22 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 23 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 24 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 25 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 26 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 27 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 28 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 29 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 30 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 31 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 32 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 33 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 34 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 35 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 36 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 37 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 38 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 39 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 40 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 41 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 42 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 43 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 44 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 45 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 46 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 47 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 48 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 49 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 50 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 51 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 52 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 53 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 54 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 55 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 56 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 57 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 58 * stride] += v_r[threadIdx.x + s + stride];
            v_r[threadIdx.x + 59 * stride] += v_r[threadIdx.x + s + stride];
        }
    } // for
}

__global__ void kmeansReduce(float *input, float *centroids, int *counter, const int n, float *v_r) {
    int Dim = 3;                                                // dim =2 cada thread vai pegar um par x,y (pular de dois em dois)
    long int i = (blockIdx.x * blockDim.x + threadIdx.x) * Dim; // i cada thread pega seu numero thread 0 pega x0y0 thread 1 pega x1y1 ...
    int pulo = gridDim.x * blockDim.x * Dim;

    // K = 4, D = 3 => 4(1+3) = 16 Variáveis K(1+D)
    float nc4_0x = 0;
    float nc4_0y = 0;
    float nc4_0z = 0;

    float nc4_1x = 0;
    float nc4_1y = 0;
    float nc4_1z = 0;

    float nc4_2x = 0;
    float nc4_2y = 0;
    float nc4_2z = 0;

    float nc4_3x = 0;
    float nc4_3y = 0;
    float nc4_3z = 0;

    float cont4_0 = 0;
    float cont4_1 = 0;
    float cont4_2 = 0;
    float cont4_3 = 0;

    // K = 5, D = 3 =: 5(1+3) = 20 Variáveis K(1+D)
    float nc5_0x = 0;
    float nc5_0y = 0;
    float nc5_0z = 0;

    float nc5_1x = 0;
    float nc5_1y = 0;
    float nc5_1z = 0;

    float nc5_2x = 0;
    float nc5_2y = 0;
    float nc5_2z = 0;

    float nc5_3x = 0;
    float nc5_3y = 0;
    float nc5_3z = 0;

    float nc5_4x = 0;
    float nc5_4y = 0;
    float nc5_4z = 0;

    float cont5_0 = 0;
    float cont5_1 = 0;
    float cont5_2 = 0;
    float cont5_3 = 0;
    float cont5_4 = 0;

    // K = 6, D = 3 => 6(1+3) = 24 Variáveis K(1+D)
    float nc6_0x = 0;
    float nc6_0y = 0;
    float nc6_0z = 0;

    float nc6_1x = 0;
    float nc6_1y = 0;
    float nc6_1z = 0;

    float nc6_2x = 0;
    float nc6_2y = 0;
    float nc6_2z = 0;

    float nc6_3x = 0;
    float nc6_3y = 0;
    float nc6_3z = 0;

    float nc6_4x = 0;
    float nc6_4y = 0;
    float nc6_4z = 0;

    float nc6_5x = 0;
    float nc6_5y = 0;
    float nc6_5z = 0;

    float cont6_0 = 0;
    float cont6_1 = 0;
    float cont6_2 = 0;
    float cont6_3 = 0;
    float cont6_4 = 0;
    float cont6_5 = 0;

    float point_d1, point_d2, point_d3;
    float k0_d1, k0_d2, k0_d3, k1_d1, k1_d2, k1_d3, k2_d1, k2_d2, k2_d3, k3_d1, k3_d2, k3_d3, k4_d1, k4_d2, k4_d3, k5_d1, k5_d2, k5_d3;
    float k;

    for (int j = 0; j < WORK; j++) {
        if (i < n) {
            // PARA K = 4
            // map
            point_d1 = input[i + 0];
            point_d2 = input[i + 1];
            point_d3 = input[i + 2];

            k0_d1 = point_d1 - centroids[0]; // primeiro centroide // ler fora pra ler uma vez só
            k0_d2 = point_d2 - centroids[1];
            k0_d3 = point_d3 - centroids[2];

            k1_d1 = point_d1 - centroids[3]; // segundo centroide
            k1_d2 = point_d2 - centroids[4];
            k1_d3 = point_d3 - centroids[5];

            k2_d1 = point_d1 - centroids[6]; // terceiro centroide
            k2_d2 = point_d2 - centroids[7];
            k2_d3 = point_d3 - centroids[8];

            k3_d1 = point_d1 - centroids[9]; // quarto centroide
            k3_d2 = point_d2 - centroids[10];
            k3_d3 = point_d3 - centroids[11];

            // k4_d1 = point_d1 - centroids[12]; // quinto centroide
            // k4_d2 = point_d2 - centroids[13];
            // k4_d3 = point_d3 - centroids[14];

            // k5_d1 = point_d1 - centroids[15]; // sexto centroide
            // k5_d2 = point_d2 - centroids[16];
            // k5_d3 = point_d3 - centroids[17];

            // dist euclidiana eleva ao quad e soma
            k0_d1 *= k0_d1;
            k0_d2 *= k0_d2;
            k0_d3 *= k0_d3;

            k1_d1 *= k1_d1;
            k1_d2 *= k1_d2;
            k1_d3 *= k1_d3;

            k2_d1 *= k2_d1;
            k2_d2 *= k2_d2;
            k2_d3 *= k2_d3;

            // reduce sum
            k0_d1 = k0_d1 + k0_d2 + k0_d3;
            k1_d2 = k1_d1 + k1_d2 + k1_d3;
            k2_d3 = k2_d1 + k2_d2 + k2_d3;

            // reduce min
            k = (k0_d1 < k1_d1) ? 0 : 1;
            k0_d1 = (k0_d1 < k1_d1) ? k0_d1 : k1_d1;
            k = (k0_d1 < k2_d1) ? k : 2;
            k0_d1 = (k0_d1 < k2_d1) ? k0_d1 : k2_d1;
            k = (k0_d1 < k3_d1) ? k : 3;

            if (k == 0) {
                nc4_0x += point_d1;
                nc4_0y += point_d2;
                nc4_0z += point_d3;
                cont4_0++;
            } else if (k == 1) {
                nc4_1x += point_d1;
                nc4_1y += point_d2;
                nc4_1z += point_d3;
                cont4_1++;
            } else if (k == 2) {
                nc4_2x += point_d1;
                nc4_2y += point_d2;
                nc4_2z += point_d3;
                cont4_2++;
            } else {
                nc4_3x += point_d1;
                nc4_3y += point_d2;
                nc4_3z += point_d3;
                cont4_3++;
            }

            // PARA K = 5
            // map
            point_d1 = input[i + 0];
            point_d2 = input[i + 1];
            point_d3 = input[i + 2];

            k0_d1 = point_d1 - centroids[12]; // primeiro centroide // ler fora pra ler uma vez só
            k0_d2 = point_d2 - centroids[13];
            k0_d3 = point_d3 - centroids[14];

            k1_d1 = point_d1 - centroids[15]; // segundo centroide
            k1_d2 = point_d2 - centroids[16];
            k1_d3 = point_d3 - centroids[17];

            k2_d1 = point_d1 - centroids[18]; // terceiro centroide
            k2_d2 = point_d2 - centroids[19];
            k2_d3 = point_d3 - centroids[20];

            k3_d1 = point_d1 - centroids[21]; // quarto centroide
            k3_d2 = point_d2 - centroids[22];
            k3_d3 = point_d3 - centroids[23];

            k4_d1 = point_d1 - centroids[24]; // quinto centroide
            k4_d2 = point_d2 - centroids[25];
            k4_d3 = point_d3 - centroids[26];

            // k5_d1 = point_d1 - centroids[15]; // sexto centroide
            // k5_d2 = point_d2 - centroids[16];
            // k5_d3 = point_d3 - centroids[17];

            // dist euclidiana eleva ao quad e soma
            k0_d1 *= k0_d1;
            k0_d2 *= k0_d2;
            k0_d3 *= k0_d3;

            k1_d1 *= k1_d1;
            k1_d2 *= k1_d2;
            k1_d3 *= k1_d3;

            k2_d1 *= k2_d1;
            k2_d2 *= k2_d2;
            k2_d3 *= k2_d3;

            k3_d1 *= k3_d1;
            k3_d2 *= k3_d2;
            k3_d3 *= k3_d3;

            k4_d1 *= k4_d1;
            k4_d2 *= k4_d2;
            k4_d3 *= k4_d3;

            // reduce sum
            k0_d1 = k0_d1 + k0_d2 + k0_d3;
            k1_d2 = k1_d1 + k1_d2 + k1_d3;
            k2_d3 = k2_d1 + k2_d2 + k2_d3;
            k3_d1 = k3_d1 + k3_d2 + k3_d3;
            k4_d1 = k4_d1 + k4_d2 + k4_d3;

            // reduce min
            k = (k0_d1 < k1_d1) ? 0 : 1;
            k0_d1 = (k0_d1 < k1_d1) ? k0_d1 : k1_d1;
            k = (k0_d1 < k2_d1) ? k : 2;
            k0_d1 = (k0_d1 < k2_d1) ? k0_d1 : k2_d1;
            k = (k0_d1 < k3_d1) ? k : 3;
            k0_d1 = (k0_d1 < k3_d1) ? k0_d1 : k3_d1;
            k = (k0_d1 < k4_d1) ? k : 4;

            if (k == 0) {
                nc5_0x += point_d1;
                nc5_0y += point_d2;
                nc5_0z += point_d3;
                cont5_0++;
            } else if (k == 1) {
                nc5_1x += point_d1;
                nc5_1y += point_d2;
                nc5_1z += point_d3;
                cont5_1++;
            } else if (k == 2) {
                nc5_2x += point_d1;
                nc5_2y += point_d2;
                nc5_2z += point_d3;
                cont5_2++;
            } else if (k == 3) {
                nc5_3x += point_d1;
                nc5_3y += point_d2;
                nc5_3z += point_d3;
                cont5_3++;
            } else {
                nc5_4x += point_d1;
                nc5_4y += point_d2;
                nc5_4z += point_d3;
                cont5_4++;
            }

            // PARA K = 6
            // map
            point_d1 = input[i + 0];
            point_d2 = input[i + 1];
            point_d3 = input[i + 2];

            k0_d1 = point_d1 - centroids[27]; // primeiro centroide // ler fora pra ler uma vez só
            k0_d2 = point_d2 - centroids[28];
            k0_d3 = point_d3 - centroids[29];

            k1_d1 = point_d1 - centroids[30]; // segundo centroide
            k1_d2 = point_d2 - centroids[31];
            k1_d3 = point_d3 - centroids[32];

            k2_d1 = point_d1 - centroids[33]; // terceiro centroide
            k2_d2 = point_d2 - centroids[34];
            k2_d3 = point_d3 - centroids[35];

            k3_d1 = point_d1 - centroids[36]; // quarto centroide
            k3_d2 = point_d2 - centroids[37];
            k3_d3 = point_d3 - centroids[38];

            k4_d1 = point_d1 - centroids[39]; // quinto centroide
            k4_d2 = point_d2 - centroids[40];
            k4_d3 = point_d3 - centroids[41];

            k5_d1 = point_d1 - centroids[42]; // sexto centroide
            k5_d2 = point_d2 - centroids[43];
            k5_d3 = point_d3 - centroids[44];

            // dist euclidiana eleva ao quad e soma
            k0_d1 *= k0_d1;
            k0_d2 *= k0_d2;
            k0_d3 *= k0_d3;

            k1_d1 *= k1_d1;
            k1_d2 *= k1_d2;
            k1_d3 *= k1_d3;

            k2_d1 *= k2_d1;
            k2_d2 *= k2_d2;
            k2_d3 *= k2_d3;

            k3_d1 *= k3_d1;
            k3_d2 *= k3_d2;
            k3_d3 *= k3_d3;

            k4_d1 *= k4_d1;
            k4_d2 *= k4_d2;
            k4_d3 *= k4_d3;

            k5_d1 *= k5_d1;
            k5_d2 *= k5_d2;
            k5_d3 *= k5_d3;

            // reduce sum
            k0_d1 = k0_d1 + k0_d2 + k0_d3;
            k1_d2 = k1_d1 + k1_d2 + k1_d3;
            k2_d3 = k2_d1 + k2_d2 + k2_d3;
            k3_d1 = k3_d1 + k3_d2 + k3_d3;
            k4_d1 = k4_d1 + k4_d2 + k4_d3;
            k5_d1 = k5_d1 + k5_d2 + k5_d3;

            // reduce min
            k = (k0_d1 < k1_d1) ? 0 : 1;
            k0_d1 = (k0_d1 < k1_d1) ? k0_d1 : k1_d1;
            k = (k0_d1 < k2_d1) ? k : 2;
            k0_d1 = (k0_d1 < k2_d1) ? k0_d1 : k2_d1;
            k = (k0_d1 < k3_d1) ? k : 3;
            k0_d1 = (k0_d1 < k3_d1) ? k0_d1 : k3_d1;
            k = (k0_d1 < k4_d1) ? k : 4;
            k0_d1 = (k0_d1 < k4_d1) ? k0_d1 : k4_d1;
            k = (k0_d1 < k5_d1) ? k : 5;

            if (k == 0) {
                nc6_0x += point_d1;
                nc6_0y += point_d2;
                nc6_0z += point_d3;
                cont6_0++;
            } else if (k == 1) {
                nc6_1x += point_d1;
                nc6_1y += point_d2;
                nc6_1z += point_d3;
                cont6_1++;
            } else if (k == 2) {
                nc6_2x += point_d1;
                nc6_2y += point_d2;
                nc6_2z += point_d3;
                cont6_2++;
            } else if (k == 3) {
                nc6_3x += point_d1;
                nc6_3y += point_d2;
                nc6_3z += point_d3;
                cont6_3++;
            } else if (k == 4) {
                nc6_4x += point_d1;
                nc6_4y += point_d2;
                nc6_4z += point_d3;
                cont6_4++;
            } else {
                nc6_5x += point_d1;
                nc6_5y += point_d2;
                nc6_5z += point_d3;
                cont6_5++;
            }
        }
        i = i + pulo;
    }

    // usando Redução na memória compartilhada
    __shared__ float partial_sum[BLOCO * 60];
    partial_sum[threadIdx.x] = nc4_0x;
    partial_sum[threadIdx.x + blockDim.x] = nc4_0y; // guarda o x de todo mundo, depois o y de todo mundo, depois o z de todo mundo, depois o cont de todos
    partial_sum[threadIdx.x + 2 * blockDim.x] = nc4_0z;
    partial_sum[threadIdx.x + 3 * blockDim.x] = nc4_1x;
    partial_sum[threadIdx.x + 4 * blockDim.x] = nc4_1y;
    partial_sum[threadIdx.x + 5 * blockDim.x] = nc4_1z;
    partial_sum[threadIdx.x + 6 * blockDim.x] = nc4_2x;
    partial_sum[threadIdx.x + 7 * blockDim.x] = nc4_2y;
    partial_sum[threadIdx.x + 8 * blockDim.x] = nc4_2z;
    partial_sum[threadIdx.x + 9 * blockDim.x] = nc4_3x;
    partial_sum[threadIdx.x + 10 * blockDim.x] = nc4_3y;
    partial_sum[threadIdx.x + 11 * blockDim.x] = nc4_3z;
    partial_sum[threadIdx.x + 12 * blockDim.x] = cont4_0;
    partial_sum[threadIdx.x + 13 * blockDim.x] = cont4_1;
    partial_sum[threadIdx.x + 14 * blockDim.x] = cont4_2;
    partial_sum[threadIdx.x + 15 * blockDim.x] = cont4_3;
    partial_sum[threadIdx.x + 16 * blockDim.x] = nc5_0x;
    partial_sum[threadIdx.x + 17 * blockDim.x] = nc5_0y;
    partial_sum[threadIdx.x + 18 * blockDim.x] = nc5_0z;
    partial_sum[threadIdx.x + 19 * blockDim.x] = nc5_1x;
    partial_sum[threadIdx.x + 20 * blockDim.x] = nc5_1y;
    partial_sum[threadIdx.x + 21 * blockDim.x] = nc5_1z;
    partial_sum[threadIdx.x + 22 * blockDim.x] = nc5_2x;
    partial_sum[threadIdx.x + 23 * blockDim.x] = nc5_2y;
    partial_sum[threadIdx.x + 24 * blockDim.x] = nc5_2z;
    partial_sum[threadIdx.x + 25 * blockDim.x] = nc5_3x;
    partial_sum[threadIdx.x + 26 * blockDim.x] = nc5_3y;
    partial_sum[threadIdx.x + 27 * blockDim.x] = nc5_3z;
    partial_sum[threadIdx.x + 28 * blockDim.x] = nc5_4x;
    partial_sum[threadIdx.x + 29 * blockDim.x] = nc5_4y;
    partial_sum[threadIdx.x + 30 * blockDim.x] = nc5_4z;
    partial_sum[threadIdx.x + 31 * blockDim.x] = cont5_0;
    partial_sum[threadIdx.x + 32 * blockDim.x] = cont5_1;
    partial_sum[threadIdx.x + 33 * blockDim.x] = cont5_2;
    partial_sum[threadIdx.x + 34 * blockDim.x] = cont5_3;
    partial_sum[threadIdx.x + 35 * blockDim.x] = cont5_4;
    partial_sum[threadIdx.x + 36 * blockDim.x] = nc6_0x;
    partial_sum[threadIdx.x + 37 * blockDim.x] = nc6_0y;
    partial_sum[threadIdx.x + 38 * blockDim.x] = nc6_0z;
    partial_sum[threadIdx.x + 39 * blockDim.x] = nc6_1x;
    partial_sum[threadIdx.x + 40 * blockDim.x] = nc6_1y;
    partial_sum[threadIdx.x + 41 * blockDim.x] = nc6_1z;
    partial_sum[threadIdx.x + 42 * blockDim.x] = nc6_2x;
    partial_sum[threadIdx.x + 43 * blockDim.x] = nc6_2y;
    partial_sum[threadIdx.x + 44 * blockDim.x] = nc6_2z;
    partial_sum[threadIdx.x + 45 * blockDim.x] = nc6_3x;
    partial_sum[threadIdx.x + 46 * blockDim.x] = nc6_3y;
    partial_sum[threadIdx.x + 47 * blockDim.x] = nc6_3z;
    partial_sum[threadIdx.x + 48 * blockDim.x] = nc6_4x;
    partial_sum[threadIdx.x + 49 * blockDim.x] = nc6_4y;
    partial_sum[threadIdx.x + 50 * blockDim.x] = nc6_4z;
    partial_sum[threadIdx.x + 51 * blockDim.x] = nc6_5x;
    partial_sum[threadIdx.x + 52 * blockDim.x] = nc6_5y;
    partial_sum[threadIdx.x + 53 * blockDim.x] = nc6_5z;
    partial_sum[threadIdx.x + 54 * blockDim.x] = cont6_0;
    partial_sum[threadIdx.x + 55 * blockDim.x] = cont6_1;
    partial_sum[threadIdx.x + 56 * blockDim.x] = cont6_2;
    partial_sum[threadIdx.x + 57 * blockDim.x] = cont6_3;
    partial_sum[threadIdx.x + 58 * blockDim.x] = cont6_4;
    partial_sum[threadIdx.x + 59 * blockDim.x] = cont6_5;

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];                                     // x0
            partial_sum[threadIdx.x + blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x];           // y0
            partial_sum[threadIdx.x + 2 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 2];   // z0
            partial_sum[threadIdx.x + 3 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 3];   // x1
            partial_sum[threadIdx.x + 4 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 4];   // y1
            partial_sum[threadIdx.x + 5 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 5];   // z1
            partial_sum[threadIdx.x + 6 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 6];   // x2
            partial_sum[threadIdx.x + 7 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 7];   // y2
            partial_sum[threadIdx.x + 8 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 8];   // z2
            partial_sum[threadIdx.x + 9 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 9];   // x3
            partial_sum[threadIdx.x + 10 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 10]; // y3
            partial_sum[threadIdx.x + 11 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 11]; // z3
            partial_sum[threadIdx.x + 12 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 12]; // c0
            partial_sum[threadIdx.x + 13 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 13]; // c1
            partial_sum[threadIdx.x + 14 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 14]; // c2
            partial_sum[threadIdx.x + 15 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 15]; // c3
            partial_sum[threadIdx.x + 16 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 16]; // x0
            partial_sum[threadIdx.x + 17 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 17]; // y0
            partial_sum[threadIdx.x + 18 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 18]; // z0
            partial_sum[threadIdx.x + 19 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 19]; // x1
            partial_sum[threadIdx.x + 20 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 20]; // y1
            partial_sum[threadIdx.x + 21 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 21]; // z1
            partial_sum[threadIdx.x + 22 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 22]; // x2
            partial_sum[threadIdx.x + 23 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 23]; // y2
            partial_sum[threadIdx.x + 24 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 24]; // z2
            partial_sum[threadIdx.x + 25 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 25]; // x3
            partial_sum[threadIdx.x + 26 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 26]; // y3
            partial_sum[threadIdx.x + 27 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 27]; // z3
            partial_sum[threadIdx.x + 28 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 28]; // x4
            partial_sum[threadIdx.x + 29 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 29]; // y4
            partial_sum[threadIdx.x + 30 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 30]; // z4
            partial_sum[threadIdx.x + 31 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 31]; // c0
            partial_sum[threadIdx.x + 32 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 32]; // c1
            partial_sum[threadIdx.x + 33 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 33]; // c2
            partial_sum[threadIdx.x + 34 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 34]; // c3
            partial_sum[threadIdx.x + 35 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 35]; // c4
            partial_sum[threadIdx.x + 36 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 36]; // x0
            partial_sum[threadIdx.x + 37 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 37]; // y0
            partial_sum[threadIdx.x + 38 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 38]; // z0
            partial_sum[threadIdx.x + 39 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 39]; // x1
            partial_sum[threadIdx.x + 40 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 40]; // y1
            partial_sum[threadIdx.x + 41 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 41]; // z1
            partial_sum[threadIdx.x + 42 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 42]; // x2
            partial_sum[threadIdx.x + 43 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 43]; // y2
            partial_sum[threadIdx.x + 44 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 44]; // z2
            partial_sum[threadIdx.x + 45 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 45]; // x3
            partial_sum[threadIdx.x + 46 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 46]; // y3
            partial_sum[threadIdx.x + 47 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 47]; // z3
            partial_sum[threadIdx.x + 48 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 48]; // x4
            partial_sum[threadIdx.x + 49 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 49]; // y4
            partial_sum[threadIdx.x + 50 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 50]; // z4
            partial_sum[threadIdx.x + 51 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 51]; // x5
            partial_sum[threadIdx.x + 52 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 52]; // y5
            partial_sum[threadIdx.x + 53 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 53]; // z5
            partial_sum[threadIdx.x + 54 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 54]; // c0
            partial_sum[threadIdx.x + 55 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 55]; // c1
            partial_sum[threadIdx.x + 56 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 56]; // c2
            partial_sum[threadIdx.x + 57 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 57]; // c3
            partial_sum[threadIdx.x + 58 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 58]; // c4
            partial_sum[threadIdx.x + 59 * blockDim.x] += partial_sum[threadIdx.x + s + blockDim.x * 59]; // c5
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        v_r[blockIdx.x] = partial_sum[0];
        v_r[blockIdx.x + gridDim.x] = partial_sum[blockDim.x];
        v_r[blockIdx.x + 2 * gridDim.x] = partial_sum[2 * blockDim.x]; // X01X02X03...Y01
        v_r[blockIdx.x + 3 * gridDim.x] = partial_sum[3 * blockDim.x];
        v_r[blockIdx.x + 4 * gridDim.x] = partial_sum[4 * blockDim.x];
        v_r[blockIdx.x + 5 * gridDim.x] = partial_sum[5 * blockDim.x];
        v_r[blockIdx.x + 6 * gridDim.x] = partial_sum[6 * blockDim.x];
        v_r[blockIdx.x + 7 * gridDim.x] = partial_sum[7 * blockDim.x];
        v_r[blockIdx.x + 8 * gridDim.x] = partial_sum[8 * blockDim.x];
        v_r[blockIdx.x + 9 * gridDim.x] = partial_sum[9 * blockDim.x];
        v_r[blockIdx.x + 10 * gridDim.x] = partial_sum[10 * blockDim.x];
        v_r[blockIdx.x + 11 * gridDim.x] = partial_sum[11 * blockDim.x];
        v_r[blockIdx.x + 12 * gridDim.x] = partial_sum[12 * blockDim.x];
        v_r[blockIdx.x + 13 * gridDim.x] = partial_sum[13 * blockDim.x];
        v_r[blockIdx.x + 14 * gridDim.x] = partial_sum[14 * blockDim.x];
        v_r[blockIdx.x + 15 * gridDim.x] = partial_sum[15 * blockDim.x];
        v_r[blockIdx.x + 16 * gridDim.x] = partial_sum[16 * blockDim.x];
        v_r[blockIdx.x + 17 * gridDim.x] = partial_sum[17 * blockDim.x];
        v_r[blockIdx.x + 18 * gridDim.x] = partial_sum[18 * blockDim.x];
        v_r[blockIdx.x + 19 * gridDim.x] = partial_sum[19 * blockDim.x];
        v_r[blockIdx.x + 20 * gridDim.x] = partial_sum[20 * blockDim.x];
        v_r[blockIdx.x + 21 * gridDim.x] = partial_sum[21 * blockDim.x];
        v_r[blockIdx.x + 22 * gridDim.x] = partial_sum[22 * blockDim.x];
        v_r[blockIdx.x + 23 * gridDim.x] = partial_sum[23 * blockDim.x];
        v_r[blockIdx.x + 24 * gridDim.x] = partial_sum[24 * blockDim.x];
        v_r[blockIdx.x + 25 * gridDim.x] = partial_sum[25 * blockDim.x];
        v_r[blockIdx.x + 26 * gridDim.x] = partial_sum[26 * blockDim.x];
        v_r[blockIdx.x + 27 * gridDim.x] = partial_sum[27 * blockDim.x];
        v_r[blockIdx.x + 28 * gridDim.x] = partial_sum[28 * blockDim.x];
        v_r[blockIdx.x + 29 * gridDim.x] = partial_sum[29 * blockDim.x];
        v_r[blockIdx.x + 30 * gridDim.x] = partial_sum[30 * blockDim.x];
        v_r[blockIdx.x + 31 * gridDim.x] = partial_sum[31 * blockDim.x];
        v_r[blockIdx.x + 32 * gridDim.x] = partial_sum[32 * blockDim.x];
        v_r[blockIdx.x + 33 * gridDim.x] = partial_sum[33 * blockDim.x];
        v_r[blockIdx.x + 34 * gridDim.x] = partial_sum[34 * blockDim.x];
        v_r[blockIdx.x + 35 * gridDim.x] = partial_sum[35 * blockDim.x];
        v_r[blockIdx.x + 36 * gridDim.x] = partial_sum[36 * blockDim.x];
        v_r[blockIdx.x + 37 * gridDim.x] = partial_sum[37 * blockDim.x];
        v_r[blockIdx.x + 38 * gridDim.x] = partial_sum[38 * blockDim.x];
        v_r[blockIdx.x + 39 * gridDim.x] = partial_sum[39 * blockDim.x];
        v_r[blockIdx.x + 40 * gridDim.x] = partial_sum[40 * blockDim.x];
        v_r[blockIdx.x + 41 * gridDim.x] = partial_sum[41 * blockDim.x];
        v_r[blockIdx.x + 42 * gridDim.x] = partial_sum[42 * blockDim.x];
        v_r[blockIdx.x + 43 * gridDim.x] = partial_sum[43 * blockDim.x];
        v_r[blockIdx.x + 44 * gridDim.x] = partial_sum[44 * blockDim.x];
        v_r[blockIdx.x + 45 * gridDim.x] = partial_sum[45 * blockDim.x];
        v_r[blockIdx.x + 46 * gridDim.x] = partial_sum[46 * blockDim.x];
        v_r[blockIdx.x + 47 * gridDim.x] = partial_sum[47 * blockDim.x];
        v_r[blockIdx.x + 48 * gridDim.x] = partial_sum[48 * blockDim.x];
        v_r[blockIdx.x + 49 * gridDim.x] = partial_sum[49 * blockDim.x];
        v_r[blockIdx.x + 50 * gridDim.x] = partial_sum[50 * blockDim.x];
        v_r[blockIdx.x + 51 * gridDim.x] = partial_sum[51 * blockDim.x];
        v_r[blockIdx.x + 52 * gridDim.x] = partial_sum[52 * blockDim.x];
        v_r[blockIdx.x + 53 * gridDim.x] = partial_sum[53 * blockDim.x];
        v_r[blockIdx.x + 54 * gridDim.x] = partial_sum[54 * blockDim.x];
        v_r[blockIdx.x + 55 * gridDim.x] = partial_sum[55 * blockDim.x];
        v_r[blockIdx.x + 56 * gridDim.x] = partial_sum[56 * blockDim.x];
        v_r[blockIdx.x + 57 * gridDim.x] = partial_sum[57 * blockDim.x];
        v_r[blockIdx.x + 58 * gridDim.x] = partial_sum[58 * blockDim.x];
        v_r[blockIdx.x + 59 * gridDim.x] = partial_sum[59 * blockDim.x];
    }
    __syncthreads();
}
